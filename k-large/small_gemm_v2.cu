//
// 64 * 64 floats
// 2 float2 per workitem
// 16 * 16 workitems per workgroup, each working on 4 floats
// this makes up to 32 * 32 floats
// 4 workgroups gets launched
//

#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void Kernel(float4* matrix_a, float4* matrix_b, float4* matrix_c) {
    unsigned tx = threadIdx.x;
    unsigned ty = threadIdx.y;
    unsigned bx = blockIdx.x;
    unsigned by = blockIdx.y;

    unsigned index_a = tx + bx * 16;
    unsigned index_b = ty + by * 16;

    unsigned index_c0 = tx + bx * 16 + ty * 32 * 4 + by * 32 * 4 * 16;
    unsigned index_c1 = index_c0 + (128/4);
    unsigned index_c2 = index_c1 + (128/4);
    unsigned index_c3 = index_c2 + (128/4);

    float4 c0 = matrix_c[index_c0];
    float4 c1 = matrix_c[index_c1];
    float4 c2 = matrix_c[index_c2];
    float4 c3 = matrix_c[index_c3];

    for(size_t i = 0; i < 128; i++) {
        float4 a = matrix_a[index_a + i * 32];
        float4 b = matrix_b[index_b + i * 32];

        c0.x += a.x * b.x;
        c0.y += a.y * b.x;
        c0.z += a.z * b.x;
        c0.w += a.w * b.x;

        c1.x += a.x * b.y;
        c1.y += a.y * b.y;
        c1.z += a.z * b.y;
        c1.w += a.w * b.y;

        c2.x += a.x * b.z;
        c2.y += a.y * b.z;
        c2.z += a.z * b.z;
        c2.w += a.w * b.z;

        c3.x += a.x * b.w;
        c3.y += a.y * b.w;
        c3.z += a.z * b.w;
        c3.w += a.w * b.w;
    }

    matrix_c[index_c0] = c0;
    matrix_c[index_c1] = c1;
    matrix_c[index_c2] = c2;
    matrix_c[index_c3] = c3;
}

int main() {
    size_t m = 128, n = 128, k = 128;
    size_t num_iter = 1024;
    size_t size = m * n * sizeof(float);
    std::vector<float> A(m * k);
    std::vector<float> B(n * k);
    std::vector<float> C(n * m);

    std::fill(A.begin(), A.end(), 1.0f);
    std::fill(B.begin(), B.end(), 2.0f);
    std::fill(C.begin(), C.end(), 1.0f);;

    float4* Ad, *Bd, *Cd;
    hipMalloc(&Ad, size);
    hipMalloc(&Bd, size);
    hipMalloc(&Cd, size);

    hipMemcpy(Ad, A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(Cd, C.data(), size, hipMemcpyHostToDevice);

    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

    for(size_t i = 0; i < num_iter; i++) {
    Kernel<<<dim3(2,2,1), dim3(16,16,1)>>>(Ad, Bd, Cd);
    }
    hipDeviceSynchronize();

    std::chrono::high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();

    double time = std::chrono::duration_cast<std::chrono::duration<double>>(stop - start).count();

    std::cout << time << std::endl;

    hipMemcpy(C.data(), Cd, size, hipMemcpyDeviceToHost);
/*
    for(size_t i = 0; i < n; i++) {
        for(size_t j = 0; j < m; j++) {
            std::cout << C[i + j * m] << " ";
        }
        std::cout << std::endl;
    }
*/
}
